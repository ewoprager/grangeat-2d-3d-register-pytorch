#include <torch/extension.h>

#include "../include/Texture.h"
#include "../include/CUDATexture.h"

namespace reg23 {

int64_t CUDATexture2D::Handle() const {
	return textureHandle;
}

int64_t CUDATexture3D::Handle() const {
	return textureHandle;
}

at::Tensor CUDATexture2D::SizeTensor() const {
	return at::tensor(size, at::dtype(at::kInt));
}

at::Tensor CUDATexture3D::SizeTensor() const {
	return at::tensor(size, at::dtype(at::kInt));
}

CUDATexture2D::CUDATexture2D(const at::Tensor &tensor, const std::string &addressModeX, const std::string &addressModeY)
	: CUDATexture2D(tensor, StringsToAddressModes<2>({{addressModeX, addressModeY}})) {
}

CUDATexture2D::CUDATexture2D(const at::Tensor &tensor, Vec<TextureAddressMode, 2> addressModes) {
	hipError_t err;

	// tensor should be a 2-dimensional array of floats on the GPU
	TORCH_CHECK(tensor.sizes().size() == 2);
	TORCH_CHECK(tensor.dtype() == at::kFloat);
	TORCH_INTERNAL_ASSERT(tensor.device().type() == at::DeviceType::CUDA);

	const at::Tensor tensorContiguous = tensor.contiguous();

	const float *const data = tensorContiguous.data_ptr<float>();

	size = Vec<int64_t, 2>::FromIntArrayRef(tensorContiguous.sizes()).Flipped();

	// Copy the given data into a CUDA array
	const hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	err = hipMallocArray(&arrayHandle, &channelDesc, size.X(), size.Y());
	if (err != hipSuccess) {
		std::cerr << "hipMallocArray failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipMallocArray failed");
	}

	err = hipMemcpy2DToArray(arrayHandle, 0, 0, data, size.X() * sizeof(float), size.X() * sizeof(float), size.Y(),
	                          hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		std::cerr << "hipMemcpy2DToArray failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipMemcpy2DToArray failed");
	}

	hipDeviceSynchronize(); // Ensure the copy is finished before tensorContiguous is destroyed

	// Create the texture object from the CUDA array
	const hipResourceDesc resourceDescriptor = {.resType = hipResourceTypeArray,
	                                             .res = {.array = {.array = arrayHandle}}};
	hipTextureDesc textureDescriptor = {.filterMode = hipFilterModeLinear, .readMode = hipReadModeElementType,
	                                     .borderColor = {0.f, 0.f, 0.f, 0.f}, .normalizedCoords = true};
	for (int i = 0; i < 2; ++i) {
		textureDescriptor.addressMode[i] = TextureAddressModeToCuda(addressModes[i]);
	}
	err = hipCreateTextureObject(&textureHandle, &resourceDescriptor, &textureDescriptor, nullptr);
	if (err != hipSuccess) {
		std::cerr << "hipCreateTextureObject failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipCreateTextureObject failed");
	}

	hipDeviceSynchronize(); // Ensure the copy is finished before tensorContiguous is destroyed
}

CUDATexture3D::CUDATexture3D(const at::Tensor &tensor, const std::string &addressModeX, const std::string &addressModeY,
                             const std::string &addressModeZ) : CUDATexture3D(
	tensor, StringsToAddressModes<3>({{addressModeX, addressModeY, addressModeZ}})) {
}

CUDATexture3D::CUDATexture3D(const at::Tensor &tensor, Vec<TextureAddressMode, 3> addressModes) {
	hipError_t err;

	// tensor should be a 3-dimensional array of floats on the GPU
	TORCH_CHECK(tensor.sizes().size() == 3);
	TORCH_CHECK(tensor.dtype() == at::kFloat);
	TORCH_INTERNAL_ASSERT(tensor.device().type() == at::DeviceType::CUDA);

	at::Tensor tensorContiguous = tensor.contiguous();

	const float *const data = tensorContiguous.data_ptr<float>();
	std::cout << "Hello, world0!\n";

	size = Vec<int64_t, 3>::FromIntArrayRef(tensorContiguous.sizes()).Flipped();
	std::cout << "Hello, world0.1!\n";

	hipExtent extent;
	extent.width = static_cast<size_t>(size.X());
	extent.height = static_cast<size_t>(size.Y());
	extent.depth = static_cast<size_t>(size.Z());

	std::cout << "Hello, world0.2!\n";

	// Copy the given data into a CUDA array
	hipChannelFormatDesc channelDesc = {};
	channelDesc.f = hipChannelFormatKindFloat;
	channelDesc.x = (int)sizeof(float) * 8;
	channelDesc.y = 0;
	channelDesc.z = 0;
	channelDesc.w = 0;
	std::cout << "Hello, world0.3!\n";

	err = hipMalloc3DArray(&arrayHandle, &channelDesc, extent);
	if (err != hipSuccess) {
		std::cerr << "hipMalloc3DArray failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipMalloc3DArray failed");
	}
	std::cout << "Hello, world0.4!\n";

	hipDeviceSynchronize(); // Ensure the copy is finished before tensorContiguous is destroyed

	std::cout << "Hello, world1!\n";

	hipMemcpy3DParms params;
	params.srcPtr = make_hipPitchedPtr((void *)data, size.X() * sizeof(float), size.X(), size.Y());
	params.dstArray = arrayHandle;
	params.extent = extent;
	params.kind = hipMemcpyDeviceToDevice;

	err = hipMemcpy3D(&params);
	if (err != hipSuccess) {
		std::cerr << "hipMemcpy3D failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipMemcpy3D failed");
	}
	std::cout << "Hello, world2!\n";

	hipDeviceSynchronize(); // Ensure the copy is finished before tensorContiguous is destroyed

	// Create the texture object from the CUDA array
	hipResourceDesc resourceDescriptor;
	resourceDescriptor.resType = hipResourceTypeArray;
	resourceDescriptor.res = {.array = {.array = arrayHandle}};

	hipTextureDesc textureDescriptor;
	textureDescriptor.filterMode = hipFilterModeLinear;
	textureDescriptor.readMode = hipReadModeElementType;
	textureDescriptor.borderColor[0] = 0.f;
	textureDescriptor.borderColor[1] = 0.f;
	textureDescriptor.borderColor[2] = 0.f;
	textureDescriptor.borderColor[3] = 0.f;
	textureDescriptor.normalizedCoords = true;
	for (int i = 0; i < 3; ++i) {
		textureDescriptor.addressMode[i] = TextureAddressModeToCuda(addressModes[i]);
	}

	err = hipCreateTextureObject(&textureHandle, &resourceDescriptor, &textureDescriptor, nullptr);
	if (err != hipSuccess) {
		std::cerr << "hipCreateTextureObject failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipCreateTextureObject failed");
	}
	std::cout << "Hello, world3!\n";

	hipDeviceSynchronize(); // Ensure the copy is finished before tensorContiguous is destroyed

	std::cout << "Hello, world4!\n";
}

} // namespace reg23