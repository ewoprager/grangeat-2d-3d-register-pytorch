#include <torch/extension.h>

#include "../include/Texture.h"
#include "../include/CUDATexture.h"

namespace reg23 {

unsigned long long CUDATexture2D::Handle() const {
	return static_cast<unsigned long long>(textureHandle);
}

unsigned long long CUDATexture3D::Handle() const {
	return static_cast<unsigned long long>(textureHandle);
}

at::Tensor CUDATexture2D::SizeTensor() const {
	return at::tensor(backingTensor.sizes(), at::dtype(at::kInt)).flip({0});
}

at::Tensor CUDATexture3D::SizeTensor() const {
	return at::tensor(backingTensor.sizes(), at::dtype(at::kInt)).flip({0});
}

CUDATexture2D::CUDATexture2D(const at::Tensor &tensor, const std::string &addressModeX, const std::string &addressModeY)
	: CUDATexture2D(tensor, StringsToAddressModes<2>({{addressModeX, addressModeY}})) {
}

CUDATexture2D::CUDATexture2D(const at::Tensor &tensor, Vec<TextureAddressMode, 2> addressModes) {
	hipError_t err;

	// tensor should be a 2-dimensional array of floats on the GPU
	TORCH_CHECK(tensor.sizes().size() == 2);
	TORCH_CHECK(tensor.dtype() == at::kFloat);
	TORCH_INTERNAL_ASSERT(tensor.device().type() == at::DeviceType::CUDA);

	backingTensor = tensor.contiguous();

	const float *const data = backingTensor.data_ptr<float>();

	const Vec<int64_t, 2> size = Vec<int64_t, 2>::FromIntArrayRef(backingTensor.sizes()).Flipped();

	// Copy the given data into a CUDA array
	const hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	err = hipMallocArray(&arrayHandle, &channelDesc, size.X(), size.Y());
	if (err != hipSuccess) {
		std::cout << "hipMallocArray failed: " << hipGetErrorString(err) << std::endl;
		throw std::bad_alloc();
	}
#ifdef DEBUG
	std::cout << "[C++] Array allocated." << std::endl;
#endif

	err = hipMemcpy2DToArray(arrayHandle, 0, 0, data, size.X() * sizeof(float), size.X() * sizeof(float), size.Y(),
	                          hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		std::cout << "hipMemcpy2DToArray failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipMemcpy2DToArray failed");
	}

	// Create the texture object from the CUDA array
	const hipResourceDesc resourceDescriptor = {.resType = hipResourceTypeArray,
	                                             .res = {.array = {.array = arrayHandle}}};
	hipTextureDesc textureDescriptor = {.filterMode = hipFilterModeLinear, .readMode = hipReadModeElementType,
	                                     .borderColor = {0.f, 0.f, 0.f, 0.f}, .normalizedCoords = true};
	for (int i = 0; i < 2; ++i) {
		textureDescriptor.addressMode[i] = TextureAddressModeToCuda(addressModes[i]);
	}
	err = hipCreateTextureObject(&textureHandle, &resourceDescriptor, &textureDescriptor, nullptr);
	if (err != hipSuccess) {
		std::cout << "hipCreateTextureObject failed: " << hipGetErrorString(err) << std::endl;
		throw std::runtime_error("hipCreateTextureObject failed");
	}
#ifdef DEBUG
	std::cout << "[C++] Texture " << static_cast<uint64_t>(textureHandle) << " created." << std::endl;
#endif

	// Ensure the tensor is no longer being used by the device before anything else can happen to it
	hipDeviceSynchronize();
}

void CUDATexture2D::CleanUp() noexcept {
#ifdef DEBUG
	std::cout << "[C++] CUDATexture2D cleaning up." << std::endl;
#endif
	hipError_t err;
	if (textureHandle) {
		err = hipDestroyTextureObject(textureHandle);
		if (err != hipSuccess) {
			std::cout << "hipDestroyTextureObject failed: " << hipGetErrorString(err) << std::endl;
			std::terminate();
		}
#ifdef DEBUG
		std::cout << "[C++] CUDA texture " << static_cast<uint64_t>(textureHandle) << " destroyed." << std::endl;
#endif
	}
	if (arrayHandle) {
		err = hipFreeArray(arrayHandle);
		if (err != hipSuccess) {
			std::cout << "hipFreeArray failed: " << hipGetErrorString(err) << std::endl;
			std::terminate();
		}
#ifdef DEBUG
		std::cout << "[C++] CUDA array freed." << std::endl;
#endif
	}
	backingTensor.reset();
}


CUDATexture3D::CUDATexture3D(const at::Tensor &tensor, const std::string &addressModeX, const std::string &addressModeY,
                             const std::string &addressModeZ) : CUDATexture3D(
	tensor, StringsToAddressModes<3>({{addressModeX, addressModeY, addressModeZ}})) {
}

CUDATexture3D::CUDATexture3D(const at::Tensor &tensor, Vec<TextureAddressMode, 3> addressModes) {
	hipError_t err;

	// tensor should be a 3-dimensional array of floats on the GPU
	TORCH_CHECK(tensor.sizes().size() == 3);
	TORCH_CHECK(tensor.dtype() == at::kFloat);
	TORCH_INTERNAL_ASSERT(tensor.device().type() == at::DeviceType::CUDA);

	backingTensor = tensor.contiguous();

	const float *const data = backingTensor.data_ptr<float>();

	const Vec<int64_t, 3> size = Vec<int64_t, 3>::FromIntArrayRef(backingTensor.sizes()).Flipped();

	hipExtent extent{};
	extent.width = static_cast<size_t>(size.X());
	extent.height = static_cast<size_t>(size.Y());
	extent.depth = static_cast<size_t>(size.Z());

	// Copy the given data into a CUDA array
	hipChannelFormatDesc channelDesc{};
	channelDesc.f = hipChannelFormatKindFloat;
	channelDesc.x = (int)sizeof(float) * 8;
	channelDesc.y = 0;
	channelDesc.z = 0;
	channelDesc.w = 0;

	err = hipMalloc3DArray(&arrayHandle, &channelDesc, extent);
	if (err != hipSuccess) {
		std::cout << "hipMalloc3DArray failed: " << hipGetErrorString(err) << std::endl << std::flush;
		throw std::bad_alloc();
	}
#ifdef DEBUG
	std::cout << "[C++] Array allocated." << std::endl << std::flush;
#endif

	hipMemcpy3DParms params{};
	params.srcPtr = make_hipPitchedPtr((void *)data, size.X() * sizeof(float), size.X(), size.Y());
	params.dstArray = arrayHandle;
	params.extent = extent;
	params.kind = hipMemcpyDeviceToDevice;

	err = hipMemcpy3D(&params);
	if (err != hipSuccess) {
		std::cout << "hipMemcpy3D failed: " << hipGetErrorString(err) << std::endl << std::flush;
		throw std::runtime_error("hipMemcpy3D failed");
	}

	// Create the texture object from the CUDA array
	hipResourceDesc resourceDescriptor{};
	resourceDescriptor.resType = hipResourceTypeArray;
	resourceDescriptor.res = {.array = {.array = arrayHandle}};

	hipTextureDesc textureDescriptor{};
	textureDescriptor.filterMode = hipFilterModeLinear;
	textureDescriptor.readMode = hipReadModeElementType;
	textureDescriptor.borderColor[0] = 0.f;
	textureDescriptor.borderColor[1] = 0.f;
	textureDescriptor.borderColor[2] = 0.f;
	textureDescriptor.borderColor[3] = 0.f;
	textureDescriptor.normalizedCoords = true;
	for (int i = 0; i < 3; ++i) {
		textureDescriptor.addressMode[i] = TextureAddressModeToCuda(addressModes[i]);
	}

	err = hipCreateTextureObject(&textureHandle, &resourceDescriptor, &textureDescriptor, nullptr);
	if (err != hipSuccess) {
		std::cout << "hipCreateTextureObject failed: " << hipGetErrorString(err) << std::endl << std::flush;
		throw std::runtime_error("hipCreateTextureObject failed");
	}
#ifdef DEBUG
	std::cout << "[C++] Texture " << static_cast<uint64_t>(textureHandle) << " created." << std::endl << std::flush;
#endif

	// Ensure the tensor is no longer being used by the device before anything else can happen to it
	hipDeviceSynchronize();
}

void CUDATexture3D::CleanUp() noexcept {
#ifdef DEBUG
	std::cout << "[C++] CUDATexture3D cleaning up." << std::endl << std::flush;
#endif
	hipError_t err;
	if (textureHandle) {
		err = hipDestroyTextureObject(textureHandle);
		if (err != hipSuccess) {
			std::cerr << "hipDestroyTextureObject failed: " << hipGetErrorString(err) << std::endl << std::flush;
			std::terminate();
		}
#ifdef DEBUG
		std::cout << "[C++] CUDA texture " << static_cast<uint64_t>(textureHandle) << " destroyed." << std::endl <<
			std::flush;
#endif
	}
	if (arrayHandle) {
		err = hipFreeArray(arrayHandle);
		if (err != hipSuccess) {
			std::cerr << "hipFreeArray failed: " << hipGetErrorString(err) << std::endl << std::flush;
			std::terminate();
		}
#ifdef DEBUG
		std::cout << "[C++] CUDA array freed." << std::endl << std::flush;
#endif
	}
	backingTensor.reset();
}

} // namespace reg23