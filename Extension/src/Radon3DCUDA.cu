#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include "../include/Texture3DCUDA.h"
#include "../include/Radon3D.h"

namespace ExtensionTest {

using CommonData = Radon3D<Texture3DCUDA>::CommonData;

__global__ void Kernel_Radon3D_CUDA(Texture3DCUDA textureIn, long numelOut, float *arrayOut,
                                    Linear<Vec<double, 3> > mappingIToOffset, const float *phiValues,
                                    const float *thetaValues, const float *rValues, long samplesPerDirection,
                                    float scaleFactor) {
	const long threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIndex >= numelOut) return;
	const Linear2<Vec<double, 3> > mappingIndexToTexCoord = Radon3D<Texture3DCUDA>::GetMappingIndexToTexCoord(
		textureIn, phiValues[threadIndex], thetaValues[threadIndex], rValues[threadIndex], mappingIToOffset);
	arrayOut[threadIndex] = scaleFactor * Radon3D<Texture3DCUDA>::IntegrateLooped(
		                        textureIn, mappingIndexToTexCoord, samplesPerDirection);
}

__host__ at::Tensor Radon3D_CUDA(const at::Tensor &volume, const at::Tensor &volumeSpacing, const at::Tensor &phiValues,
                                 const at::Tensor &thetaValues, const at::Tensor &rValues, long samplesPerDirection) {
	CommonData common = Radon3D<Texture3DCUDA>::Common(volume, volumeSpacing, phiValues, thetaValues, rValues,
	                                                   samplesPerDirection, at::DeviceType::CUDA);

	float *resultFlatPtr = common.flatOutput.data_ptr<float>();

	const at::Tensor phiFlatContiguous = phiValues.flatten().contiguous();
	const float *phiFlatPtr = phiFlatContiguous.data_ptr<float>();
	const at::Tensor thetaFlatContiguous = thetaValues.flatten().contiguous();
	const float *thetaFlatPtr = thetaFlatContiguous.data_ptr<float>();
	const at::Tensor rFlatContiguous = rValues.flatten().contiguous();
	const float *rFlatPtr = rFlatContiguous.data_ptr<float>();

	constexpr int blockSize = 512;
	const int gridSize = (static_cast<unsigned>(common.flatOutput.numel()) + blockSize - 1) / blockSize;
	Kernel_Radon3D_CUDA<<<gridSize, blockSize>>>(std::move(common.inputTexture), common.flatOutput.numel(),
	                                             resultFlatPtr, common.mappingIndexToOffset, phiFlatPtr, thetaFlatPtr,
	                                             rFlatPtr, samplesPerDirection, common.scaleFactor);
	return common.flatOutput.view(phiValues.sizes());
}

__global__ void Kernel_DRadon3DDR_CUDA(Texture3DCUDA textureIn, long numelOut, float *arrayOut,
                                       Linear<Vec<double, 3> > mappingIToOffset, const float *phiValues,
                                       const float *thetaValues, const float *rValues, long samplesPerDirection,
                                       float scaleFactor) {
	const long threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIndex >= numelOut) return;
	const float phi = phiValues[threadIndex];
	const float theta = thetaValues[threadIndex];
	const float r = rValues[threadIndex];
	const Linear2<Vec<double, 3> > mappingIndexToTexCoord = Radon3D<Texture3DCUDA>::GetMappingIndexToTexCoord(
		textureIn, phi, theta, r, mappingIToOffset);
	const Vec<double, 3> dTexCoordDR = Radon3D<Texture3DCUDA>::GetDTexCoordDR(textureIn, phi, theta, r);
	arrayOut[threadIndex] = scaleFactor * Radon3D<Texture3DCUDA>::DIntegrateLoopedDMappingParameter(
		                        textureIn, mappingIndexToTexCoord, dTexCoordDR, samplesPerDirection);
}

__host__ at::Tensor DRadon3DDR_CUDA(const at::Tensor &volume, const at::Tensor &volumeSpacing,
                                    const at::Tensor &phiValues, const at::Tensor &thetaValues,
                                    const at::Tensor &rValues, long samplesPerDirection) {
	CommonData common = Radon3D<Texture3DCUDA>::Common(volume, volumeSpacing, phiValues, thetaValues, rValues,
	                                                   samplesPerDirection, at::DeviceType::CUDA);

	float *resultFlatPtr = common.flatOutput.data_ptr<float>();

	const at::Tensor phiFlatContiguous = phiValues.flatten().contiguous();
	const float *phiFlatPtr = phiFlatContiguous.data_ptr<float>();
	const at::Tensor thetaFlatContiguous = thetaValues.flatten().contiguous();
	const float *thetaFlatPtr = thetaFlatContiguous.data_ptr<float>();
	const at::Tensor rFlatContiguous = rValues.flatten().contiguous();
	const float *rFlatPtr = rFlatContiguous.data_ptr<float>();

	constexpr int blockSize = 512;
	const int gridSize = (static_cast<unsigned>(common.flatOutput.numel()) + blockSize - 1) / blockSize;
	Kernel_DRadon3DDR_CUDA<<<gridSize, blockSize>>>(std::move(common.inputTexture), common.flatOutput.numel(),
	                                                resultFlatPtr, common.mappingIndexToOffset, phiFlatPtr,
	                                                thetaFlatPtr, rFlatPtr, samplesPerDirection, common.scaleFactor);
	return common.flatOutput.view(phiValues.sizes());
}

struct Radon3DV2Consts {
	hipTextureObject_t textureHandle{};
	long samplesPerDirection{};
	double scaleFactor{};
	float *patchSumsArray{};
};

__device__ __constant__ Radon3DV2Consts radon3DV2Consts{};

__global__ void Kernel_Radon3D_CUDA_V2(Linear2<Vec<double, 3> > mappingIndexToTexCoord) {
	extern __shared__ float buffer[];

	const long i = blockIdx.x * blockDim.x + threadIdx.x;
	const long j = blockIdx.y * blockDim.y + threadIdx.y;
	const long localIndex = threadIdx.y * blockDim.x + threadIdx.x;
	if (i >= radon3DV2Consts.samplesPerDirection || j >= radon3DV2Consts.samplesPerDirection) {
		buffer[localIndex] = 0.f;
		return;
	}

	const Vec<double, 3> texCoord = mappingIndexToTexCoord(Vec<double, 3>::Full(static_cast<double>(i)),
	                                                       Vec<double, 3>::Full(static_cast<double>(j)));
	buffer[localIndex] = tex3D<float>(radon3DV2Consts.textureHandle, texCoord.X(), texCoord.Y(), texCoord.Z());

	__syncthreads();

	for (long cutoff = (blockDim.x * blockDim.y) / 2; cutoff > 0; cutoff /= 2) {
		if (localIndex < cutoff) {
			buffer[localIndex] += buffer[localIndex + cutoff];
		}

		__syncthreads();
	}
	if (localIndex == 0) {
		radon3DV2Consts.patchSumsArray[blockIdx.y * gridDim.x + blockIdx.x] = radon3DV2Consts.scaleFactor * buffer[0];
	}
}

__global__ void Kernel_Radon3D_CUDA_V3(Linear2<Vec<double, 3> > mappingIndexToTexCoord) {
	extern __shared__ float buffer[];

	// REQUIRED: blockDim.x must be equal to blockDim.y

	const long i = blockIdx.x * blockDim.x + threadIdx.x;
	const long j = blockIdx.y * blockDim.y + threadIdx.y;
	// const long localIndex = threadIdx.y * blockDim.x + threadIdx.x;
	if (i >= radon3DV2Consts.samplesPerDirection || j >= radon3DV2Consts.samplesPerDirection) {
		buffer[threadIdx.y * blockDim.x + threadIdx.x] = 0.f;
		return;
	}

	const Vec<double, 3> texCoord = mappingIndexToTexCoord(Vec<double, 3>::Full(static_cast<double>(i)),
	                                                       Vec<double, 3>::Full(static_cast<double>(j)));
	buffer[threadIdx.y * blockDim.x + threadIdx.x] = tex3D<float>(radon3DV2Consts.textureHandle, texCoord.X(),
	                                                              texCoord.Y(), texCoord.Z());

	__syncthreads();

	for (long cutoff = blockDim.x / 2; cutoff > 0; cutoff /= 2) {
		if (threadIdx.x < cutoff && threadIdx.y < cutoff) {
			buffer[threadIdx.y * blockDim.x + threadIdx.x] += buffer[threadIdx.y * blockDim.x + threadIdx.x + cutoff] +
				buffer[(threadIdx.y + cutoff) * blockDim.x + threadIdx.x] + buffer[
					(threadIdx.y + cutoff) * blockDim.x + threadIdx.x + cutoff];
		}

		__syncthreads();
	}

	if (threadIdx.x == 0 && threadIdx.y) {
		radon3DV2Consts.patchSumsArray[blockIdx.y * gridDim.x + blockIdx.x] = radon3DV2Consts.scaleFactor * buffer[0];
	}
}

__host__ at::Tensor Radon3D_CUDA_V2(const at::Tensor &volume, const at::Tensor &volumeSpacing,
                                    const at::Tensor &phiValues, const at::Tensor &thetaValues,
                                    const at::Tensor &rValues, long samplesPerDirection) {
	CommonData common = Radon3D<Texture3DCUDA>::Common(volume, volumeSpacing, phiValues, thetaValues, rValues,
	                                                   samplesPerDirection, at::DeviceType::CUDA);

	const at::Tensor phiFlat = phiValues.flatten();
	const at::Tensor thetaFlat = thetaValues.flatten();
	const at::Tensor rFlat = rValues.flatten();

	constexpr dim3 blockSize = {32, 32};
	constexpr size_t bufferSize = blockSize.x * blockSize.y * sizeof(float);
	const dim3 gridSize = {(static_cast<unsigned>(samplesPerDirection) + blockSize.x - 1) / blockSize.x,
	                       (static_cast<unsigned>(samplesPerDirection) + blockSize.y - 1) / blockSize.y};
	const at::Tensor patchSums = torch::zeros(at::IntArrayRef({gridSize.y, gridSize.x}), common.flatOutput.options());
	float *patchSumsPtr = patchSums.data_ptr<float>();

	Radon3DV2Consts constants{common.inputTexture.GetHandle(), samplesPerDirection, common.scaleFactor, patchSumsPtr};
	CudaMemcpyToObjectSymbol(radon3DV2Consts, constants);

	for (long i = 0; i < common.flatOutput.numel(); ++i) {
		const Linear2<Vec<double, 3> > mappingIndexToTexCoord = Radon3D<Texture3DCUDA>::GetMappingIndexToTexCoord(
			common.inputTexture, phiFlat[i].item().toFloat(), thetaFlat[i].item().toFloat(), rFlat[i].item().toFloat(),
			common.mappingIndexToOffset);

		Kernel_Radon3D_CUDA_V3<<<gridSize, blockSize, bufferSize>>>(mappingIndexToTexCoord);

		common.flatOutput.index_put_({i}, patchSums.sum());
	}
	return common.flatOutput.view(phiValues.sizes());
}

struct DRadon3DDRV2Consts {
	hipTextureObject_t textureHandle{};
	long samplesPerDirection{};
	double scaleFactor{};
	float *patchSumsArray{};
	long volumeWidth{};
	long volumeHeight{};
	long volumeDepth{};
};

__device__ __constant__ DRadon3DDRV2Consts dRadon3DDRV2Consts{};

__global__ void Kernel_DRadon3DDR_CUDA_V2(Linear2<Vec<double, 3> > mappingIndexToTexCoord, Vec<double, 3> dTexCoordDR) {
	extern __shared__ float buffer[];

	// REQUIRED: blockDim.x must be equal to blockDim.y

	const long i = blockIdx.x * blockDim.x + threadIdx.x;
	const long j = blockIdx.y * blockDim.y + threadIdx.y;
	// const long localIndex = threadIdx.y * blockDim.x + threadIdx.x;
	if (i >= dRadon3DDRV2Consts.samplesPerDirection || j >= dRadon3DDRV2Consts.samplesPerDirection) {
		buffer[threadIdx.y * blockDim.x + threadIdx.x] = 0.f;
		return;
	}

	const Vec<double, 3> texCoord = mappingIndexToTexCoord(Vec<double, 3>::Full(static_cast<double>(i)),
	                                                       Vec<double, 3>::Full(static_cast<double>(j)));
	buffer[threadIdx.y * blockDim.x + threadIdx.x] =
		Texture3DCUDA::DSampleDX(dRadon3DDRV2Consts.volumeWidth, dRadon3DDRV2Consts.textureHandle, texCoord) *
		dTexCoordDR.X() +
		Texture3DCUDA::DSampleDY(dRadon3DDRV2Consts.volumeHeight, dRadon3DDRV2Consts.textureHandle, texCoord) *
		dTexCoordDR.Y() + Texture3DCUDA::DSampleDZ(dRadon3DDRV2Consts.volumeDepth, dRadon3DDRV2Consts.textureHandle,
		                                           texCoord) * dTexCoordDR.Z();

	__syncthreads();

	for (long cutoff = blockDim.x / 2; cutoff > 0; cutoff /= 2) {
		if (threadIdx.x < cutoff && threadIdx.y < cutoff) {
			buffer[threadIdx.y * blockDim.x + threadIdx.x] += buffer[threadIdx.y * blockDim.x + threadIdx.x + cutoff] +
				buffer[(threadIdx.y + cutoff) * blockDim.x + threadIdx.x] + buffer[
					(threadIdx.y + cutoff) * blockDim.x + threadIdx.x + cutoff];
		}

		__syncthreads();
	}

	if (threadIdx.x == 0 && threadIdx.y) {
		dRadon3DDRV2Consts.patchSumsArray[blockIdx.y * gridDim.x + blockIdx.x] =
			dRadon3DDRV2Consts.scaleFactor * buffer[0];
	}
}

__host__ at::Tensor DRadon3DDR_CUDA_V2(const at::Tensor &volume, const at::Tensor &volumeSpacing,
                                       const at::Tensor &phiValues, const at::Tensor &thetaValues,
                                       const at::Tensor &rValues, long samplesPerDirection) {
	CommonData common = Radon3D<Texture3DCUDA>::Common(volume, volumeSpacing, phiValues, thetaValues, rValues,
	                                                   samplesPerDirection, at::DeviceType::CUDA);

	const at::Tensor phiFlat = phiValues.flatten();
	const at::Tensor thetaFlat = thetaValues.flatten();
	const at::Tensor rFlat = rValues.flatten();

	constexpr dim3 blockSize = {32, 32};
	constexpr size_t bufferSize = blockSize.x * blockSize.y * sizeof(float);
	const dim3 gridSize = {(static_cast<unsigned>(samplesPerDirection) + blockSize.x - 1) / blockSize.x,
	                       (static_cast<unsigned>(samplesPerDirection) + blockSize.y - 1) / blockSize.y};
	const at::Tensor patchSums = torch::zeros(at::IntArrayRef({gridSize.y, gridSize.x}), common.flatOutput.options());
	float *patchSumsPtr = patchSums.data_ptr<float>();

	DRadon3DDRV2Consts constants{common.inputTexture.GetHandle(), samplesPerDirection, common.scaleFactor, patchSumsPtr,
	                             volume.sizes()[2], volume.sizes()[1], volume.sizes()[0]};
	CudaMemcpyToObjectSymbol(dRadon3DDRV2Consts, constants);

	for (long i = 0; i < common.flatOutput.numel(); ++i) {
		const float phi = phiFlat[i].item().toFloat();
		const float theta = thetaFlat[i].item().toFloat();
		const float r = rFlat[i].item().toFloat();
		const Linear2<Vec<double, 3> > mappingIndexToTexCoord = Radon3D<Texture3DCUDA>::GetMappingIndexToTexCoord(
			common.inputTexture, phi, theta, r, common.mappingIndexToOffset);
		const Vec<double, 3> dTexCoordDR = Radon3D<Texture3DCUDA>::GetDTexCoordDR(common.inputTexture, phi, theta, r);

		Kernel_DRadon3DDR_CUDA_V2<<<gridSize, blockSize, bufferSize>>>(mappingIndexToTexCoord, dTexCoordDR);

		common.flatOutput.index_put_({i}, patchSums.sum());
	}
	return common.flatOutput.view(phiValues.sizes());
}

} // namespace ExtensionTest