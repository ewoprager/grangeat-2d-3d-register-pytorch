#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include "../include/ResampleSinogram3D.h"
#include "../include/SinogramClassic3DCPU.h"
#include "../include/SinogramHEALPixCPU.h"

namespace reg23 {

using CommonData = ResampleSinogram3D::CommonData;
using ConstantGeometry = ResampleSinogram3D::ConstantGeometry;

template <typename sinogram_t> __global__ void Kernel_ResampleSinogram3D_CUDA(
	sinogram_t inputSinogram, const ConstantGeometry geometry, const float *phiValues, const float *rValues,
	long numelOut, float *resultPtr) {

	const long threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadIndex >= numelOut) return;

	const float phi = phiValues[threadIndex];
	const float r = rValues[threadIndex];
	resultPtr[threadIndex] = ResampleSinogram3D::ResamplePlane(inputSinogram, geometry, phi, r);
}

/**
 * @brief
 *
 *	Note: Assumes that the projection matrix projects onto the x-y plane, and that the radial coordinates (phi, r)
 *	in that plane measure phi right-hand rule about the z-axis from the positive x-direction
 *
 * @param sinogram3d
 * @param sinogramType
 * @param rSpacing
 * @param projectionMatrix
 * @param phiValues
 * @param rValues
 * @return
 */
__host__ at::Tensor ResampleSinogram3D_CUDA(const at::Tensor &sinogram3d, const std::string &sinogramType,
                                            double rSpacing, const at::Tensor &projectionMatrix,
                                            const at::Tensor &phiValues, const at::Tensor &rValues) {
	CommonData common = ResampleSinogram3D::Common(sinogram3d, sinogramType, projectionMatrix, phiValues, rValues,
	                                               at::DeviceType::CUDA);

	const at::Tensor phiFlatContiguous = phiValues.flatten().contiguous();
	const float *phiFlatPtr = phiFlatContiguous.data_ptr<float>();
	const at::Tensor rFlatContiguous = rValues.flatten().contiguous();
	const float *rFlatPtr = rFlatContiguous.data_ptr<float>();

	float *resultFlatPtr = common.flatOutput.data_ptr<float>();

	switch (common.sinogramType) {
	case ResampleSinogram3D::SinogramType::CLASSIC: {
		const SinogramClassic3DCPU sinogram = SinogramClassic3DCPU::FromTensor(sinogram3d, rSpacing);
		int minGridSize, blockSize;
		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		                                   &Kernel_ResampleSinogram3D_CUDA<SinogramClassic3DCPU>, 0, 0);
		const int gridSize = (static_cast<unsigned>(common.flatOutput.numel()) + blockSize - 1) / blockSize;

		Kernel_ResampleSinogram3D_CUDA<SinogramClassic3DCPU><<<gridSize, blockSize>>>(
			sinogram, common.geometry, phiFlatPtr, rFlatPtr, common.flatOutput.numel(), resultFlatPtr);
		break;
	}
	case ResampleSinogram3D::SinogramType::HEALPIX: {
		const SinogramHEALPixCPU sinogram = SinogramHEALPixCPU::FromTensor(sinogram3d, rSpacing);
		int minGridSize, blockSize;
		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
		                                   &Kernel_ResampleSinogram3D_CUDA<SinogramHEALPixCPU>, 0, 0);
		const int gridSize = (static_cast<unsigned>(common.flatOutput.numel()) + blockSize - 1) / blockSize;

		Kernel_ResampleSinogram3D_CUDA<SinogramHEALPixCPU><<<gridSize, blockSize>>>(
			sinogram, common.geometry, phiFlatPtr, rFlatPtr, common.flatOutput.numel(), resultFlatPtr);
		break;
	}
	}
	return common.flatOutput.view(phiValues.sizes());
}

} // namespace reg23